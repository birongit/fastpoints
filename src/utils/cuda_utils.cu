#include "cuda_utils.h"
#include "../../../../../usr/local/cuda/include/hip/hip_runtime.h"

int get_cuda_version() {

    int version;
    hipError_t cuda_err = hipRuntimeGetVersion(&version);

    return (cuda_err == hipSuccess) ? version : 0;
}

int get_driver_version() {

    int version;
    hipError_t cuda_err = hipDriverGetVersion(&version);

    return (cuda_err == hipSuccess) ? version : 0;
}