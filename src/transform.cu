#include "hip/hip_runtime.h"
#include "../src/point_cloud.h"

#include "transform.h"

#include <hip/hip_runtime_api.h>
#include <>

__global__ void shifting(PointXYZI *d_points, double *d_shift) {

    int idx = threadIdx.x;
    d_points[idx].x += d_shift[0];
    d_points[idx].y += d_shift[1];
    d_points[idx].z += d_shift[2];
}

__global__ void rotate(PointXYZI *d_points, Quaternion *d_quaternion) {

    int idx = threadIdx.x;

    Quaternion q(d_points[idx]);

    Quaternion q_prime = *d_quaternion * q * d_quaternion->inverse();

    d_points[idx].x = q_prime.x;
    d_points[idx].y = q_prime.y;
    d_points[idx].z = q_prime.z;

}


PointCloud ShiftPoints(PointCloud &h_cloud, std::vector<double> shift) {

    PointXYZI *d_points;
    double *d_shift;

    PointXYZI *h_points = &(h_cloud.points[0]);
    double *h_shift = &(shift[0]);

    hipMalloc((void**) &d_points, h_cloud.points.size() * sizeof(PointXYZI));
    hipMalloc((void**) &d_shift, shift.size() * sizeof(double));

    hipMemcpy(d_points, h_points, h_cloud.points.size() * sizeof(PointXYZI), hipMemcpyHostToDevice);
    hipMemcpy(d_shift, h_shift, shift.size() * sizeof(double), hipMemcpyHostToDevice);

    int N = h_cloud.points.size();

    shifting<<<1,N>>>(d_points, d_shift);

    hipMemcpy(h_points, d_points, h_cloud.points.size() * sizeof(PointXYZI), hipMemcpyDeviceToHost);
    hipMemcpy(h_shift, d_shift, shift.size() * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_points);
    hipFree(d_shift);

    return h_cloud;
}

PointCloud RotatePoints(PointCloud &h_cloud, Quaternion &quaternion){

    PointXYZI *d_points;
    PointXYZI *h_points = &(h_cloud.points[0]);

    hipMalloc((void**) &d_points, h_cloud.points.size() * sizeof(PointXYZI));

    hipMemcpy(d_points, h_points, h_cloud.points.size() * sizeof(PointXYZI), hipMemcpyHostToDevice);

    int N = h_cloud.points.size();

    rotate<<<1,N>>>(d_points, &quaternion);

    hipMemcpy(h_points, d_points, h_cloud.points.size() * sizeof(PointXYZI), hipMemcpyDeviceToHost);

    hipFree(d_points);

    return h_cloud;

}