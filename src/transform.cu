#include "hip/hip_runtime.h"
#include "../src/point_cloud.h"

#include "transform.h"

#include <hip/hip_runtime_api.h>
#include <>

__global__ void shifting(PointXYZI *d_points, double *d_shift) {

    int idx = threadIdx.x;
    d_points[idx].x += d_shift[0];
    d_points[idx].y += d_shift[1];
    d_points[idx].z += d_shift[2];
}

PointCloud shiftPoints(PointCloud &h_cloud, std::vector<double> shift) {

    PointXYZI *d_points;
    double *d_shift;

    PointXYZI *h_points = &(h_cloud.points[0]);
    double *h_shift = &(shift[0]);

    hipMalloc((void**) &d_points, h_cloud.points.size() * sizeof(PointXYZI));
    hipMalloc((void**) &d_shift, shift.size() * sizeof(double));

    hipMemcpy(d_points, h_points, h_cloud.points.size() * sizeof(PointXYZI), hipMemcpyHostToDevice);
    hipMemcpy(d_shift, h_shift, shift.size() * sizeof(double), hipMemcpyHostToDevice);

    int N = h_cloud.points.size();

    shifting<<<1,N>>>(d_points, d_shift);

    hipMemcpy(h_points, d_points, h_cloud.points.size() * sizeof(PointXYZI), hipMemcpyDeviceToHost);
    hipMemcpy(h_shift, d_shift, shift.size() * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_points);
    hipFree(d_shift);

    return h_cloud;
}