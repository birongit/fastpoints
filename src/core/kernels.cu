#include "hip/hip_runtime.h"
#include "kernels.h"

typedef double (*func)(double n, double m);

__global__ void reduce(const Point3D * const d_points, Point3D *d_result, func* f, Point3D *d_reduce) {


    const int idx = threadIdx.x;


    d_reduce[idx] = d_points[idx];
    d_reduce[idx] = d_points[idx];

    __syncthreads();

    Point3D res;

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (idx < s)
        {
            d_reduce[idx].x = (*f)(d_reduce[idx].x, d_reduce[idx + s].x);
            d_reduce[idx].y = (*f)(d_reduce[idx].y, d_reduce[idx + s].y);
            d_reduce[idx].z = (*f)(d_reduce[idx].z, d_reduce[idx + s].z);
        }
        __syncthreads();
    }

    if (idx == 0) {
        *d_result = d_reduce[idx];
    }

}

__device__ double max_d(double n, double m) {
    return (m > n) ? m : n;
}
__device__ func max_func = max_d;

void reduce_max(PointCloud<Point3D> &cloud) {

    Point3D *d_points;
    Point3D *d_result;
    Point3D *d_reduce;

    func* h_f;
    func* d_f;
    h_f = (func*)malloc(sizeof(func));
    hipMalloc((void**)&d_f,sizeof(func));
    hipMemcpyFromSymbol( &h_f[0], HIP_SYMBOL(max_func), sizeof(func));
    hipMemcpy(d_f,h_f,sizeof(func),hipMemcpyHostToDevice);

    hipMalloc(&d_reduce, sizeof(Point3D) * cloud.points.size());
    hipMalloc((void**) &d_result, sizeof(Point3D));
    hipMalloc((void**) &d_points, sizeof(Point3D) * cloud.points.size());
    hipMemcpy(d_points, &cloud.points[0], sizeof(Point3D) * cloud.points.size(), hipMemcpyHostToDevice);

    int N = cloud.points.size();

    reduce<<<1,N>>>(d_points, d_result, d_f, d_reduce);

    hipDeviceSynchronize();

    Point3D h_result;
    hipMemcpy(&h_result, d_result, sizeof(Point3D), hipMemcpyDeviceToHost);

    std::cout << "Result of kernel: " << h_result.x << " " << h_result.y << " " << h_result.z << std::endl;

    hipFree(d_points);
    hipFree(d_result);
    hipFree(d_reduce);

    hipFree(d_f);
    free(h_f);

}